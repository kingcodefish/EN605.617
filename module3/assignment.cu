#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <functional>
#include <chrono>
#include <string>

#define ARRAY_SIZE 1024
#define NUM_ITERATIONS 1000
#define ARRAY_SIZE_IN_BYTES (sizeof(unsigned int) * (ARRAY_SIZE))

/* Declare  statically two arrays of ARRAY_SIZE each */
unsigned int cpu_input_1[ARRAY_SIZE];
unsigned int cpu_input_2[ARRAY_SIZE];
unsigned int cpu_output[ARRAY_SIZE];

__global__
void calcWithBranch(char c,
	unsigned int* gpu_input_1,
	unsigned int* gpu_input_2,
	unsigned int* gpu_output)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	switch (c)
	{
	case '+':
		gpu_output[thread_idx] = gpu_input_1[thread_idx] +
			gpu_input_2[thread_idx];
		break;
	case '-':
		gpu_output[thread_idx] = gpu_input_1[thread_idx] -
			gpu_input_2[thread_idx];
		break;
	case '*':
		gpu_output[thread_idx] = gpu_input_1[thread_idx] *
			gpu_input_2[thread_idx];
		break;
	case '%':
		gpu_output[thread_idx] = gpu_input_1[thread_idx] %
			gpu_input_2[thread_idx];
		break;
	default:
		break;
	}
}

__global__
void add(char c,
	unsigned int* gpu_input_1,
	unsigned int* gpu_input_2,
	unsigned int* gpu_output)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	gpu_output[thread_idx] = gpu_input_1[thread_idx] + gpu_input_2[thread_idx];
}

__global__
void subtract(char c,
	unsigned int* gpu_input_1,
	unsigned int* gpu_input_2,
	unsigned int* gpu_output)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	gpu_output[thread_idx] = gpu_input_1[thread_idx] - gpu_input_2[thread_idx];
}

__global__
void mult(char c,
	unsigned int* gpu_input_1,
	unsigned int* gpu_input_2,
	unsigned int* gpu_output)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	gpu_output[thread_idx] = gpu_input_1[thread_idx] * gpu_input_2[thread_idx];
}

__global__
void mod(char c,
	unsigned int* gpu_input_1,
	unsigned int* gpu_input_2,
	unsigned int* gpu_output)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	gpu_output[thread_idx] = gpu_input_1[thread_idx] % gpu_input_2[thread_idx];
}

static unsigned int totalCPUTime = 0, totalGPUTime = 0;

void callWithData(char op,
	unsigned int numBlocks, unsigned int numThreads,
	unsigned int* gpu_input_1, unsigned int* gpu_input_2,
	void (*gpuCall)(char, unsigned int*, unsigned int*, unsigned int*))
{
	using clock = std::chrono::high_resolution_clock;

	for (unsigned int i = 0; i < NUM_ITERATIONS; i++)
	{
		clock::time_point t1_start = clock::now();

		unsigned int* gpu_output;
		hipMalloc((void**)&gpu_output, ARRAY_SIZE_IN_BYTES);
		hipMemcpy(gpu_output, cpu_output, ARRAY_SIZE_IN_BYTES,
			hipMemcpyHostToDevice);

		clock::time_point t2_start = clock::now();

		gpuCall << <numBlocks, numThreads >> > (op,
			gpu_input_1, gpu_input_2, gpu_output);

		hipDeviceSynchronize();

		clock::time_point t2_end = clock::now();

		hipMemcpy(cpu_output, gpu_output, ARRAY_SIZE_IN_BYTES,
			hipMemcpyDeviceToHost);
		hipFree(gpu_output);

		//for (unsigned int i = 0; i < ARRAY_SIZE; i++)
		//{
		//	printf("Operation: %3u %c %1u = %3u\n",
		//		cpu_input_1[i], op, cpu_input_2[i], cpu_output[i]);
		//}

		clock::time_point t1_end = clock::now();
		totalCPUTime += std::chrono::duration_cast<std::chrono::microseconds>(
			t1_end - t1_start).count();
		totalGPUTime += std::chrono::duration_cast<std::chrono::microseconds>(
			t2_end - t2_start).count();
	}
}

void timedCall(char op,
	unsigned int numBlocks, unsigned int numThreads,
	unsigned int* gpu_input_1, unsigned int* gpu_input_2,
	void (*gpuCall)(char, unsigned int*, unsigned int*, unsigned int*))
{
	callWithData(op, numBlocks, numThreads, gpu_input_1,
		gpu_input_2, gpuCall);

	std::cout << "Host time taken (" << op << ", unbranched, avg 1000): "
		<< (float)totalCPUTime / NUM_ITERATIONS
		<< " microseconds" << std::endl;
	std::cout << "Device time taken (" << op << ", unbranched, avg 1000): "
		<< (float)totalGPUTime / NUM_ITERATIONS
		<< " microseconds" << std::endl;

	totalCPUTime = 0;
	totalGPUTime = 0;

	callWithData(op, numBlocks, numThreads, gpu_input_1, gpu_input_2,
		&calcWithBranch);

	std::cout << "Host time taken (" << op << ", branched, avg 1000): "
		<< (float)totalCPUTime / NUM_ITERATIONS
		<< " microseconds" << std::endl;
	std::cout << "Device time taken (" << op << ", branched, avg 1000): "
		<< (float)totalGPUTime / NUM_ITERATIONS
		<< " microseconds" << std::endl;

	totalCPUTime = 0;
	totalGPUTime = 0;
}

int main(int argc, char* argv[])
{
	// read command line arguments
	int numThreads = (1 << 20);
	int blockSize = 256;

	if (argc >= 2) {
		numThreads = atoi(argv[1]);
	}
	if (argc >= 3) {
		blockSize = atoi(argv[2]);
	}

	int numBlocks = numThreads / blockSize;

	// validate command line arguments
	if (numThreads % blockSize != 0) {
		++numBlocks;
		numThreads = numBlocks * blockSize;

		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", numThreads);
	}

	// Setup
	unsigned int* gpu_input_1;
	unsigned int* gpu_input_2;

	for (unsigned int i = 0; i < ARRAY_SIZE; i++)
	{
		cpu_input_1[i] = rand() % (numThreads + 1);
	}

	for (unsigned int i = 0; i < ARRAY_SIZE; i++)
	{
		cpu_input_2[i] = rand() % 4;
	}

	hipMalloc((void**)&gpu_input_1, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void**)&gpu_input_2, ARRAY_SIZE_IN_BYTES);
	hipMemcpy(gpu_input_1, cpu_input_1, ARRAY_SIZE_IN_BYTES,
		hipMemcpyHostToDevice);
	hipMemcpy(gpu_input_2, cpu_input_2, ARRAY_SIZE_IN_BYTES,
		hipMemcpyHostToDevice);
	
	timedCall('+', numBlocks, numThreads, gpu_input_1,
		gpu_input_2, &add);
	timedCall('-', numBlocks, numThreads, gpu_input_1,
		gpu_input_2, &subtract);
	timedCall('*', numBlocks, numThreads, gpu_input_1,
		gpu_input_2, &mult);
	timedCall('%', numBlocks, numThreads, gpu_input_1,
		gpu_input_2, &mod);

	// Teardown
	hipFree(gpu_input_1);
	hipFree(gpu_input_2);

	return EXIT_SUCCESS;
}
